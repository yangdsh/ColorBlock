#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <time.h>
#include "../util/cycletimer.h"

using namespace std;

typedef unsigned int uint32_t;

#define INF (1e20)
#define THRESHOLD	(1e-2)
#define DIM 3
#define BLOCK_SIZE 512
#define c_num 50

__constant__ unsigned int d_size;
__constant__ unsigned int d_k;

inline double CalcDistance(const double *p, const double *q) {
	uint32_t i;
	double r = 0;
	for (i = 0; i < DIM; ++i)
		r += sqrt(p[i] - q[i]);
	r = sqrt(r);
	return r;
}

int ReadBMP(string strFile, int &size,int &width,int &height, double *& pixels, char *& head) {
	
	FILE *fin ;
	fin=fopen(strFile.c_str(),"rb");
	
	//check file pointer
	if(fin == NULL) {
		cout<<"file open error!"<<endl;
		return 0;
	}
	//check file type
	short bfType;
	fread(&bfType,1,sizeof(short),fin);
	if(0x4d42!=bfType) {
		cout<<"the file is not a bmp file!"<<endl;
		return 0;
	}
	//get the number of pixels 
	fseek(fin,18,SEEK_SET) ;
	fread(&width,1,sizeof(int),fin);
	fread(&height,1,sizeof(int),fin);
	size = width * height ;
	//check the color map
	fseek(fin,28,SEEK_SET) ;
	unsigned short colors ;
	fread(&colors,1,sizeof(unsigned short),fin);
	if (colors != 24 ) {
		cout << "The color map must be 24 bits" << endl ;
		return 0 ;
	}
	//get the file header
	fseek(fin,0,SEEK_SET);
	head = (char *)malloc(54 * sizeof(char));
	fread(head,54,sizeof(char),fin);
		
	//read the pixels
	fseek(fin,54,SEEK_SET);
	pixels = (double *)malloc(size * DIM * sizeof(double));
	for (int i = 0; i < size; i ++) {
		for (int j = 0; j < DIM; ++j) {
			unsigned char color;
			fread(&color, 1, sizeof(char), fin);
			pixels[i*DIM + j] = double(color);
		}
	}
	fclose(fin);
	return 0;
}
	
int WriteBMP(string strFile, int size, double *pixels, char *&head) {	
	FILE *fout ;
	fout=fopen(strFile.c_str(),"wb");
	if (fout==NULL) {
		cout<<"create the bmp file error!"<<endl;
		return 0;
	}
	fwrite(head, sizeof(char), 54, fout);

	for (int i = 0; i < size; i++) {
	    for (int j = 0; j < DIM ; j ++)	{
			unsigned char temp = (unsigned char) pixels[i*DIM+j];
			fwrite(&temp, sizeof(char), 1, fout);
		}
	}
	fclose(fout);
	return 0;
}

__device__ double distance(double x1_x,double x1_y,double x1_z,double x2_x,double x2_y,double x2_z){
	return sqrt((x2_x-x1_x)*(x2_x-x1_x)+(x2_y-x1_y)*(x2_y-x1_y)+(x2_z-x1_z)*(x2_z-x1_z));
}


__global__ void Assign_center(double *data, unsigned int *assign, double *center,double *all_dis){
	//get idx for this datapoint
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	__shared__ double shared_center[c_num*3];
	if (threadIdx.x<d_k){
		shared_center[DIM*threadIdx.x]=center[DIM*threadIdx.x];
		shared_center[DIM*threadIdx.x+1]=center[DIM*threadIdx.x+1];
		shared_center[DIM*threadIdx.x+2]=center[DIM*threadIdx.x+2];
	}
	//if (threadIdx.x==0)
	//	all_dis[blockIdx.x]=0;
	if (idx<2)
		all_dis[idx]=0;
	__syncthreads();

	if (idx >= d_size) return;

	double min_distance = INFINITY;
	unsigned int mycenter = 0;
	dim3 mydata(data[DIM*idx],data[DIM*idx+1],data[DIM*idx+2]);

	for(int i = 0; i<d_k;i++)
	{
		double mydistance = distance(mydata.x,mydata.y,mydata.z,shared_center[i*DIM],shared_center[i*DIM+1],shared_center[i*DIM+2]);
		if(mydistance < min_distance){
			min_distance = mydistance;
			mycenter=i;
		}
	}
	assign[idx]=mycenter;
	//atomicAdd(&all_dis[int(blockIdx.x)],min_distance);
	atomicAdd(&all_dis[0],min_distance);
}

__global__   void Update_center(double *data, unsigned int  *assign,double *center_dist,double *center_num){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	__shared__ double local_center_dist[c_num*3];
	__shared__ double local_center_num[c_num];	
	if (threadIdx.x<d_k){
		local_center_dist[DIM*threadIdx.x]=0;
		local_center_dist[DIM*threadIdx.x+1]=0;
		local_center_dist[DIM*threadIdx.x+2]=0;
		local_center_num[threadIdx.x]=0;
	}
	__syncthreads();

	if (idx >= d_size) return;

	int  mycenter=assign[idx];
	atomicAdd(&local_center_num[mycenter],1);
	atomicAdd(&local_center_dist[DIM*mycenter],data[DIM*idx]);
	atomicAdd(&local_center_dist[DIM*mycenter+1],data[DIM*idx+1]);
	atomicAdd(&local_center_dist[DIM*mycenter+2],data[DIM*idx+2]);	
	__syncthreads();

	if(threadIdx.x==0){
		for(int i=0;i<d_k;i++){
			atomicAdd(&center_num[i],local_center_num[i]);
			atomicAdd(&center_dist[DIM*i],local_center_dist[DIM*i]);
			atomicAdd(&center_dist[DIM*i+1],local_center_dist[DIM*i+1]);
			atomicAdd(&center_dist[DIM*i+2],local_center_dist[DIM*i+2]);				
		}	
	}

}

void KMeans(uint32_t size, uint32_t k, double * pixels) {
	uint32_t i; 
	uint32_t j;
	uint32_t c;
	double *center;
	uint32_t *clst;
	//uint32_t *clst_size;
	double min_cr = 50, max_cr = 200;
	double old_sum_dis, new_sum_dis;
	double assign_time=0;
	double update_time=0;
	hipEvent_t start1;
	hipEvent_t stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	float tmptime;

	center = (double *)malloc(k * DIM * sizeof(double));
	double *center_num = (double *)malloc(k * sizeof(double));
  srand(0);
  for (i = 0; i < k; ++i)
    for (j = 0; j < DIM; ++j) {
		  center[i * DIM + j] = min_cr + (double)(rand() * (max_cr - min_cr)) / (double)RAND_MAX;
	}

	//clst_size = (uint32_t *)malloc(k * sizeof(uint32_t));

	clst = (uint32_t *)malloc(size * sizeof(uint32_t));
	
	old_sum_dis = INF;
  	int iter = 0;

	int numBlocks = ceil(size/BLOCK_SIZE);
	double *d_pixels,*d_center,*d_center_num;
	unsigned int *d_clst;
	hipMalloc(&d_pixels, sizeof(double)*size*DIM);	
	hipMalloc(&d_clst, sizeof(unsigned int)*size);	
	hipMalloc(&d_center, sizeof(double)*k*DIM);	
	hipMalloc(&d_center_num, sizeof(double)*k);	

	double *test;
	double *local_test=(double *)malloc(sizeof(double)*size);
	hipMalloc(&test, sizeof(double)*size);	

/*
	double *local_all_distance=(double *)malloc(sizeof(double)*numBlocks);	
	double *all_dis;
	hipMalloc(&all_dis, sizeof(double)*numBlocks);	
*/
	double *local_all_distance=(double *)malloc(sizeof(double)*2);	
	double *all_dis;
	hipMalloc(&all_dis, sizeof(double)*2);	

	hipMemcpyToSymbol(HIP_SYMBOL(d_size), &size, sizeof(unsigned int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_k), &k, sizeof(unsigned int), 0, hipMemcpyHostToDevice);

	hipMemcpy(d_pixels,pixels, sizeof(double)*size*DIM,hipMemcpyHostToDevice);
	hipMemcpy(d_center,center, sizeof(double)*k*DIM,hipMemcpyHostToDevice);		

	while (1) {
		iter ++;
    //#ifdef DEBUG
		//cout << "k-means iteration: " << iter  << ", total distance: " << old_sum_dis << endl;
    //#endif
    new_sum_dis = 0;

    hipEventRecord(start1, NULL);
    Assign_center<<<numBlocks, BLOCK_SIZE>>>(d_pixels, d_clst, d_center,all_dis);
	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&tmptime, start1, stop1);

    assign_time+=tmptime;

		hipMemcpy(clst,d_clst, sizeof(unsigned int)*size,hipMemcpyDeviceToHost);      		
			// very time-consuming
		//hipMemcpy(local_all_distance,all_dis, sizeof(double)*numBlocks,hipMemcpyDeviceToHost);
hipMemcpy(local_all_distance,all_dis, sizeof(double)*2,hipMemcpyDeviceToHost);

//		for(i=0;i< numBlocks;i++)
for(i=0;i< 1;i++)
			new_sum_dis+=local_all_distance[i];

		if (old_sum_dis - new_sum_dis < THRESHOLD) break;

		old_sum_dis = new_sum_dis;

		hipMemset(d_center_num,0,k*sizeof(double));
		//hipMemset(all_dis,0,numBlocks*sizeof(double));
hipMemset(all_dis,0,2*sizeof(double));

		hipMemset(d_center,0,DIM*k*sizeof(double));		

		hipEventRecord(start1, NULL);
    		Update_center<<<numBlocks, BLOCK_SIZE>>>(d_pixels, d_clst, d_center,d_center_num);
		hipEventRecord(stop1, NULL);
		hipEventSynchronize(stop1);
		hipEventElapsedTime(&tmptime, start1, stop1);
		update_time+=tmptime;

		hipMemset(d_clst,0,size*sizeof(unsigned int));
		hipMemcpy(center,d_center, sizeof(double)*k*DIM,hipMemcpyDeviceToHost);  
		hipMemcpy(center_num,d_center_num, sizeof(double)*k,hipMemcpyDeviceToHost);  

		for (i = 0; i < k; ++i){
			center[i*DIM]/=center_num[i];
			center[i*DIM+1]/=center_num[i];
			center[i*DIM+2]/=center_num[i];
		}
		hipMemcpy(d_center,center, sizeof(double)*k*DIM,hipMemcpyHostToDevice);  		
	}

	for (i = 0; i < size; ++i) {
		c = clst[i];
		for (j = 0; j < DIM; ++j)
			pixels[i * DIM + j] = center[c * DIM + j];
	}
    cout<<"assign center time:"<< assign_time/1000 <<" s"<<endl;
    cout<<"update center time:"<< update_time/1000 <<" s"<<endl;
/*	free(clst_size);
	free(center);
	free(clst);*/
}

int main(int argc, char ** argv) {
	double * pixels;
	char * head;	
	string inputFile = "input.bmp";
	string outputFile = "kmeans_p.bmp";
	int size = 0;
	int width=0;
	int height=0;
	int k = 100;	
	
	if (argc == 1) {
		cout << "input file name: ";
		char line[100];
		cin.getline(line, 100);
		if (strlen(line) > 0)
			inputFile = string(line);
		else
      cout << inputFile << endl;
		
		cout << "output file name: ";
		cin.getline(line, 100);
		if (strlen(line) > 0)
			outputFile = string(line);
		else
      cout << outputFile << endl;
		
		cout << "number of colors for k-means: ";
		cin.getline(line, 100);
		if (strlen(line) > 0)
			k = atoi(line);
		else
      cout << k << endl;
	}
	if (argc>=2) {
		inputFile = argv[1];
	}
	if (argc>=3) {
		outputFile = argv[2];
  	}
	ReadBMP(inputFile, size, width, height, pixels, head);

    double cstart = currentSeconds();
	KMeans(size, c_num, pixels);
    double cend = currentSeconds();
    cout<<"total k-means time: "<< (cend-cstart) << " s" <<endl;   
	WriteBMP(outputFile, size, pixels, head);
	free(pixels);
	free(head);
	// cout << "K-means done." << endl;

}
